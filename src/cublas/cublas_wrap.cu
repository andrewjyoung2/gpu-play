#include "hip/hip_runtime.h"
#include <cassert>
#include <limits>
#include "src/cublas/cublas_wrap.hpp"

namespace cublas_wrap {

__host__ float Ddot(const std::vector<double>& A, const std::vector<double>& B)
{
  assert(A.size() == B.size());

  hipblasHandle_t cublasH { nullptr };
  hipStream_t   stream  { nullptr };

  double result = std::numeric_limits<double>::min();

  double* d_A { nullptr };
  double* d_B { nullptr };

  // creat cuBLAS handle and bind to stream
  CUBLAS_CHECK(hipblasCreate(&cublasH));
  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  // Allocate device memory, copy to device
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A),
                        sizeof(double) * A.size()));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B),
                        sizeof(double) * B.size()));

  CUDA_CHECK(hipMemcpyAsync(d_A,
                             A.data(),
                             sizeof(double) * A.size(),
                             hipMemcpyHostToDevice,
                             stream));
  CUDA_CHECK(hipMemcpyAsync(d_B,
                             B.data(),
                             sizeof(double) * B.size(),
                             hipMemcpyHostToDevice,
                             stream));
  // Compute the dot product
  const int incx = 1;
  const int incy = 1;
  CUBLAS_CHECK(hipblasDdot(cublasH, A.size(), d_A, incx, d_B, incy, &result));

  // Synchronize stream to copy result to host
  CUDA_CHECK(hipStreamSynchronize(stream));

  // Cleanup
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));

  CUBLAS_CHECK(hipblasDestroy(cublasH));

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipDeviceReset());

  return result;
}

} // namespace cublas_wrap

