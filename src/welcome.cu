#include "hip/hip_runtime.h"
#include <iostream>
#include "src/welcome.hpp"

namespace welcome {

__global__ void welcome(char* msg) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    msg[idx] = d_message[idx];
}

__host__ std::vector<char> execute_kernel(const std::string& msg)
{
  const size_t length = msg.size() + 1;

  // Copy message to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_message), msg.c_str(), length * sizeof(char));
  
  // Allocate device memory
  char* d_msg { nullptr };
  hipMalloc(&d_msg, length * sizeof(char)); // TODO: error handling

  // Launch welcome kernel
  welcome<<<1, length>>>(d_msg);
    
  // Copy result back to host
  std::vector<char> result(length);
  hipMemcpy(result.data(),
             d_msg,
             length * sizeof(char),
             hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_msg);

  return result;
}

} // namespace welcome

