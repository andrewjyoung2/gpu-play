
#include <hip/hip_runtime.h>
#include <cassert>
#include <stdexcept>

namespace math {

static const size_t MAX_CUDA_THREADS { 256 };

//------------------------------------------------------------------------------
__global__ void VectorMultiply(float* C,
                               float* A,
                               float* B)
{
  const int idx = threadIdx.x;

  C[idx] = A[idx] * B[idx];
}

//------------------------------------------------------------------------------
__global__ void Accumulate(float* d_result, float* d_A, const size_t len)
{
  __shared__ float scratch[MAX_CUDA_THREADS];

  const int idx     = threadIdx.x;

  size_t numThreads = len;
  float* readPtr    = d_A;
  float* readEnd    = d_A + len;
  float* writePtr   = scratch;

  do {
    numThreads = (numThreads + 1) >> 1;

    if (idx < numThreads) {
      writePtr[idx] = readPtr[2 * idx];

      if (readPtr + 2 * idx + 1 < readEnd) {
        writePtr[idx] += readPtr[2 * idx + 1];
      }

      readPtr = writePtr;
      readEnd = readPtr + numThreads;
      writePtr += numThreads;
    }

    __syncthreads();

  } while (1 != numThreads);

  if (0 == idx) {
    *d_result = *readPtr;
  }
}

//------------------------------------------------------------------------------
__host__ void AccumulateDevice(float* d_result, float* d_A, const size_t len)
{
  // TODO: how much scratch is actually necessary?
  if (2 * len >= MAX_CUDA_THREADS) {
    throw std::runtime_error("len must be less than MAX_CUDA_THREADS");
  }

  Accumulate<<<1, (len >> 1) >>>(d_result, d_A, len);
}

//------------------------------------------------------------------------------
__host__ float AccumulateHost(float* A, const size_t len)
{
  if (len >= MAX_CUDA_THREADS) {
    throw std::runtime_error("len must be less than MAX_CUDA_THREADS");
  }

  const size_t bytes = len * sizeof(float);

  // Allocate device memory
  float* d_A { nullptr };
  if (hipSuccess != hipMalloc(&d_A, bytes)) {
    throw std::runtime_error("hipMalloc failed");
  }
  float* d_result { nullptr };
  if (hipSuccess != hipMalloc(&d_result, sizeof(float))) {
    throw std::runtime_error("hipMalloc failed");
  }

  // Copy inputs to device
  if (hipSuccess != hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice)) {
    throw std::runtime_error("Failed transfer from host to device");
  }

  AccumulateDevice(d_result, d_A, len);

  // Copy result to host
  float result { 0.7734f };
  if (hipSuccess != hipMemcpy(&result,
                                d_result,
                                sizeof(float),
                                hipMemcpyDeviceToHost)) {
    throw std::runtime_error("Failed transfer from device to host");
  }

  // Cleanup
  hipFree(d_A);
  hipFree(d_result);

  return result;
}

//------------------------------------------------------------------------------
__host__ void VectorMultiplyDevice(float* C,
                                   float* A,
                                   float* B,
                                   const size_t len)
{
  if (len >= MAX_CUDA_THREADS) {
    throw std::runtime_error("len must be less than MAX_CUDA_THREADS");
  }

  VectorMultiply<<<1, len>>>(C, A, B);
}

//------------------------------------------------------------------------------
__host__ void VectorMultiplyHost(float* C,
                                 float* A,
                                 float* B,
                                 const size_t len)
{
  const size_t bytes = len * sizeof(float);

  float* d_C { nullptr };
  float* d_A { nullptr };
  float* d_B { nullptr };

  // Allocate device memory
  if (hipSuccess != hipMalloc(&d_C, bytes)) {
    throw std::runtime_error("hipMalloc failed");
  }
  if (hipSuccess != hipMalloc(&d_A, bytes)) {
    throw std::runtime_error("hipMalloc failed");
  }
  if (hipSuccess != hipMalloc(&d_B, bytes)) {
    throw std::runtime_error("hipMalloc failed");
  }

  // Copy inputs to device
  if (hipSuccess != hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice)) {
    throw std::runtime_error("Failed transfer from host to device");
  }
  if (hipSuccess != hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice)) {
    throw std::runtime_error("Failed transfer from host to device");
  }

  // Execute kernel
  VectorMultiplyDevice(d_C, d_A, d_B, len);

  // Copy output to host
  if (hipSuccess != hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost)) {
    throw std::runtime_error("Failed transfer from device to host");
  }

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

} // namespace math

