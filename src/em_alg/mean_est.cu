#include "hip/hip_runtime.h"
#include <chrono>
#include "src/common/assert.hpp"
#include "src/em_alg/mean_est.hpp"

namespace EM { namespace CUDA {

//------------------------------------------------------------------------------
// TODO: __global__ void MeanEstKernel

//------------------------------------------------------------------------------
__host__ void MeanEstHost(common::Matrix<float>&       means,
                          const common::Matrix<float>& posteriors,
                          const common::Matrix<float>& observations)
{
  const int numClasses = means.rows();
  const int dimension  = means.cols();
  const int numObs     = posteriors.cols();

  ASSERT(dimension == 2); // limit of the current implementation

  ASSERT(posteriors.rows()    == numClasses);
  ASSERT(observations.rows() == numObs);
  ASSERT(observations.cols() == dimension);

  // Allocate device memory
  float* d_means        { nullptr };
  float* d_posteriors   { nullptr };
  float* d_observations { nullptr };

  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_means),
             means.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_posteriors),
             posteriors.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_observations),
             observations.size() * sizeof(float)));

  // Transfer data from host to device
  auto start = std::chrono::high_resolution_clock::now();

  CUDA_CHECK(hipMemcpy(d_observations,
                        observations.data(),
                        observations.size() * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_posteriors,
                        posteriors.data(),
                        posteriors.size() * sizeof(float),
                        hipMemcpyHostToDevice));

  auto end      = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to execute EM::CUDA::PosteriorDevice = " << duration.count()
            << " microseconds"                          << std::endl;

  // Run the calculation
  start = std::chrono::high_resolution_clock::now();

  MeanEstDevice(d_means,
                d_posteriors,
                d_observations,
                dimension,
                numClasses,
                numObs);

  end      = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to execute EM::CUDA::MeanEstDevice = " << duration.count()
            << " microseconds"                              << std::endl;

  // Transfer results from device to host
  start = std::chrono::high_resolution_clock::now();

  CUDA_CHECK(hipMemcpy(means.data(),
                        d_means,
                        means.size() * sizeof(float),
                        hipMemcpyDeviceToHost));

  end      = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to transfer from device to host = " << duration.count()
            << " microseconds"                           << std::endl;

  // Cleanup
  CUDA_CHECK(hipFree(d_means));
  CUDA_CHECK(hipFree(d_posteriors));
  CUDA_CHECK(hipFree(d_observations));

  CUDA_CHECK(hipDeviceReset());
}

__host__ void MeanEstDevice(float*    d_means,
                            float*    d_posteriors,
                            float*    d_observations,
                            const int dimension,
                            const int numClasses,
                            const int numObs)
{
  ASSERT(nullptr != d_means);
  ASSERT(nullptr != d_posteriors);
  ASSERT(nullptr != d_observations);
}

} // namespace CUDA
} // namespace EM
