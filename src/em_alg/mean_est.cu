#include "hip/hip_runtime.h"
#include <chrono>
#include "src/common/assert.hpp"
#include "src/em_alg/mean_est.hpp"

namespace EM { namespace CUDA {

//------------------------------------------------------------------------------
__global__ void MeanEstKernel(float*    d_means,
                              float*    d_posteriors,
                              float*    d_observations,
                              const int dimension,
                              const int numClasses,
                              const int numObs)
{
  const int j = threadIdx.x; // column index, coordinate index
  const int n = threadIdx.y; // row index, class index

  // denominator = sum_k posterior(j, k)
  // numerator   = sum_k posterior(j, k) * obs(k, n)
  float denom { 0 };
  float num   { 0 };
  for (int k = 0; k < numObs; ++k) {
    denom += d_posteriors[k + j * numObs];
    num   += d_posteriors[k + j * numObs] * d_observations[n + k * dimension];
  }

  // write result to entry (j, n) of means matrix
  d_means[n + j * dimension] = num / denom;
}

//------------------------------------------------------------------------------
__host__ void MeanEstHost(common::Matrix<float>&       means,
                          const common::Matrix<float>& posteriors,
                          const common::Matrix<float>& observations)
{
  const int numClasses = means.rows();
  const int dimension  = means.cols();
  const int numObs     = posteriors.cols();

  ASSERT(dimension == 2); // limit of the current implementation

  ASSERT(posteriors.rows()   == numClasses);
  ASSERT(observations.rows() == numObs);
  ASSERT(observations.cols() == dimension);

  // TODO: remove
  ASSERT(numClasses == 3);
  ASSERT(numObs     == 500);

  // Allocate device memory
  float* d_means        { nullptr };
  float* d_posteriors   { nullptr };
  float* d_observations { nullptr };

  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_means),
             means.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_posteriors),
             posteriors.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_observations),
             observations.size() * sizeof(float)));

  // Transfer data from host to device
  auto start = std::chrono::high_resolution_clock::now();

  CUDA_CHECK(hipMemcpy(d_observations,
                        observations.data(),
                        observations.size() * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_posteriors,
                        posteriors.data(),
                        posteriors.size() * sizeof(float),
                        hipMemcpyHostToDevice));

  auto end      = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to execute EM::CUDA::PosteriorDevice = " << duration.count()
            << " microseconds"                          << std::endl;

  // Run the calculation
  start = std::chrono::high_resolution_clock::now();

  MeanEstDevice(d_means,
                d_posteriors,
                d_observations,
                dimension,
                numClasses,
                numObs);

  hipDeviceSynchronize();

  end      = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to execute EM::CUDA::MeanEstDevice = " << duration.count()
            << " microseconds"                              << std::endl;

  // Transfer results from device to host
  start = std::chrono::high_resolution_clock::now();

  CUDA_CHECK(hipMemcpy(means.data(),
                        d_means,
                        means.size() * sizeof(float),
                        hipMemcpyDeviceToHost));

  end      = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Time to transfer from device to host = " << duration.count()
            << " microseconds"                           << std::endl;

  // Cleanup
  CUDA_CHECK(hipFree(d_means));
  CUDA_CHECK(hipFree(d_posteriors));
  CUDA_CHECK(hipFree(d_observations));

  CUDA_CHECK(hipDeviceReset());
}

__host__ void MeanEstDevice(float*    d_means,
                            float*    d_posteriors,
                            float*    d_observations,
                            const int dimension,
                            const int numClasses,
                            const int numObs)
{
  ASSERT(nullptr != d_means);
  ASSERT(nullptr != d_posteriors);
  ASSERT(nullptr != d_observations);

  // Run kernel
  const int xDim = numClasses;
  const int yDim = dimension;
  ASSERT(xDim * yDim < 256);

  const dim3 threadsPerBlock(xDim, yDim);
  const int  numBlocks { 1 };

  MeanEstKernel<<<numBlocks, threadsPerBlock>>>(d_means,
                                                d_posteriors,
                                                d_observations,
                                                dimension,
                                                numClasses,
                                                numObs);
}

} // namespace CUDA
} // namespace EM
